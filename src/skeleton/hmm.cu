#include "hip/hip_runtime.h"

/////////////////////////////
// GENERAL IMPLEMENTATIONS //
/////////////////////////////

#ifdef PRECOMPUTE_PREDECESSORS
__device__
void forward_prob_predecessors(
    const prob_t *alpha_prev, int instance, state_t state, prob_t *probs,
    HMM_DECL_PARAMS) {
  state_t *predecessors = predecessor_table + state;
  for (state_t i = 0; i < NUM_PREDS; i++) {
    state_t pred = predecessors[i * NUM_STATES];
    probs[i] =
      alpha_prev[instance * NUM_STATES + pred] + transition_prob(pred, state, HMM_CALL_ARGS);
  }
}

__device__
void viterbi_max_predecessor(
    const prob_t *chi_prev, int instance, state_t state, state_t *maxs,
    prob_t *maxp, HMM_DECL_PARAMS) {
  state_t *predecessors = predecessor_table + state;
  for (state_t i = 0; i < NUM_PREDS; i++) {
    state_t pred = predecessors[i * NUM_STATES];
    prob_t p = chi_prev[instance * NUM_STATES + pred] + transition_prob(pred, state, HMM_CALL_ARGS);
    if (p > *maxp) {
      *maxs = pred;
      *maxp = p;
    }
  }
}
#endif

const prob_t inf = 1.0 / 0.0;

extern "C"
__global__
void forward_init(
    const obs_t* __restrict__ obs, int maxlen, prob_t* __restrict__ alpha_zero,
    HMM_DECL_PARAMS) {
  state_t state = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int instance = blockIdx.y;
  if (state < NUM_STATES) {
    obs_t x = obs[instance * maxlen];
    alpha_zero[instance * NUM_STATES + state] =
      init_prob(state, HMM_CALL_ARGS) + output_prob(state, x, HMM_CALL_ARGS);
  }
}

__device__
prob_t log_sum_exp(const prob_t* probs) {
  prob_t maxp = probs[0];
  for (int i = 1; i < NUM_PREDS; i++) {
    if (probs[i] > maxp) maxp = probs[i];
  }
  if (maxp == -inf) return maxp;
  prob_t sum = 0.0;
  for (int i = 0; i < NUM_PREDS; i++) {
    sum += expf(probs[i] - maxp);
  }
  return maxp + logf(sum);
}

extern "C"
__global__ void forward_step(
    const obs_t* __restrict__ obs, const int* __restrict__ obs_lens, int maxlen,
    const prob_t* __restrict__ alpha_prev, prob_t* __restrict__ alpha_curr,
#ifdef PRECOMPUTE_PREDECESSORS
    int t, prob_t* __restrict__ probs_table, HMM_DECL_PARAMS) {
#else
    int t, HMM_DECL_PARAMS) {
#endif
  state_t state = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int instance = blockIdx.y;
  if (state < NUM_STATES) {
    size_t idx = instance * NUM_STATES + state;
    if (t < obs_lens[instance]) {
      obs_t x = obs[instance * maxlen + t];
#ifdef PRECOMPUTE_PREDECESSORS
      prob_t *probs = probs_table + instance * NUM_STATES * NUM_PREDS + state * NUM_PREDS;
      forward_prob_predecessors(alpha_prev, instance, state, probs, HMM_CALL_ARGS);
#else
      prob_t probs[NUM_PREDS];
      int pidx = forward_prob_predecessors(alpha_prev, instance, state, probs, HMM_CALL_ARGS);
      while (pidx < NUM_PREDS) probs[pidx++] = -inf;
#endif
      alpha_curr[idx] = log_sum_exp(probs) + output_prob(state, x, HMM_CALL_ARGS);
    } else if (t == obs_lens[instance]) {
      // We only need to copy the alpha data once - past this point, both alpha
      // vectors will contain the same data.
      alpha_curr[idx] = alpha_prev[idx];
    }
  }
}

__device__
void forward_max_warp_reduce(volatile prob_t *maxp, unsigned int tid) {
  if (maxp[tid + 32] > maxp[tid]) {
    maxp[tid] = maxp[tid + 32];
  }
  if (maxp[tid + 16] > maxp[tid]) {
    maxp[tid] = maxp[tid + 16];
  }
  if (maxp[tid + 8] > maxp[tid]) {
    maxp[tid] = maxp[tid + 8];
  }
  if (maxp[tid + 4] > maxp[tid]) {
    maxp[tid] = maxp[tid + 4];
  }
  if (maxp[tid + 2] > maxp[tid]) {
    maxp[tid] = maxp[tid + 2];
  }
  if (maxp[tid + 1] > maxp[tid]) {
    maxp[tid] = maxp[tid + 1];
  }
}

extern "C"
__global__
void forward_max(
    const prob_t* __restrict__ alpha, prob_t* __restrict__ result) {
  unsigned int idx = threadIdx.x;
  unsigned int instance = blockIdx.x;
  unsigned int lo = instance * NUM_STATES;

  __shared__ prob_t maxp[512];
  if (idx < NUM_STATES) {
    maxp[idx] = alpha[lo + idx];
  } else {
    maxp[idx] = -inf;
  }
  for (int i = lo + idx + 512; i < lo + NUM_STATES; i += 512) {
    if (alpha[i] > maxp[idx]) {
      maxp[idx] = alpha[i];
    }
  }
  __syncthreads();
  
  if (idx < 256) {
    if (maxp[idx + 256] > maxp[idx]) {
      maxp[idx] = maxp[idx + 256];
    }
  }
  __syncthreads();
  if (idx < 128) {
    if (maxp[idx + 128] > maxp[idx]) {
      maxp[idx] = maxp[idx + 128];
    }
  }
  __syncthreads();
  if (idx < 64) {
    if (maxp[idx + 64] > maxp[idx]) {
      maxp[idx] = maxp[idx + 64];
    }
  }
  __syncthreads();
  if (idx < 32) forward_max_warp_reduce(maxp, idx);

  if (idx == 0) {
    result[instance] = maxp[0];
  }
}

__device__
void forward_sum_warp_reduce(volatile prob_t *psum, unsigned int tid) {
  psum[tid] = psum[tid] + psum[tid + 32];
  psum[tid] = psum[tid] + psum[tid + 16];
  psum[tid] = psum[tid] + psum[tid + 8];
  psum[tid] = psum[tid] + psum[tid + 4];
  psum[tid] = psum[tid] + psum[tid + 2];
  psum[tid] = psum[tid] + psum[tid + 1];
}

extern "C"
__global__
void forward_log_sum_exp(
    const prob_t* __restrict__ alpha, prob_t* __restrict__ result) {
  unsigned int idx = threadIdx.x;
  unsigned int instance = blockIdx.x;
  unsigned int lo = instance * NUM_STATES;

  // Retrieve the maximum value for the current instance, as computed in the
  // max kernel.
  prob_t maxp = result[instance];

  __shared__ prob_t psum[512];
  if (idx < NUM_STATES) {
    psum[idx] = expf(alpha[lo + idx] - maxp);
  } else {
    psum[idx] = 0.0;
  }
  for (int i = lo + idx + 512; i < lo + NUM_STATES; i += 512) {
    psum[idx] = psum[idx] + expf(alpha[i] - maxp);
  }
  __syncthreads();

  // Compute the sum of all these exponents
  if (idx < 256) psum[idx] = psum[idx] + psum[idx + 256];
  __syncthreads();
  if (idx < 128) psum[idx] = psum[idx] + psum[idx + 128];
  __syncthreads();
  if (idx < 64) psum[idx] = psum[idx] + psum[idx + 64];
  __syncthreads();
  if (idx < 32) forward_sum_warp_reduce(psum, idx);

  // The first thread of each block writes the result
  if (idx == 0) {
    result[instance] = maxp + logf(psum[0]);
  }
}

extern "C"
__global__
void viterbi_init(
    const obs_t* __restrict__ obs, int maxlen, prob_t* __restrict__ chi_zero,
    HMM_DECL_PARAMS) {
  state_t state = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int instance = blockIdx.y;
  if (state < NUM_STATES) {
    obs_t x = obs[instance * maxlen];
    chi_zero[instance * NUM_STATES + state] =
      init_prob(state, HMM_CALL_ARGS) + output_prob(state, x, HMM_CALL_ARGS);
  }
}

extern "C"
__global__
void viterbi_init_batch(
    const obs_t* __restrict__ obs, const int* __restrict__ obs_lens, int maxlen,
    const state_t* __restrict__ seq, prob_t* __restrict__ chi_zero, int t,
    HMM_DECL_PARAMS) {
  state_t state = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int instance = blockIdx.y;
  if (state < NUM_STATES) {
    if (t < obs_lens[instance]) {
      obs_t x = obs[instance * maxlen + t];
      state_t last_state = seq[instance * maxlen + t - 1];
      if (state == last_state) {
        chi_zero[instance * NUM_STATES + state] = output_prob(state, x, HMM_CALL_ARGS);
      } else {
        chi_zero[instance * NUM_STATES + state] = -inf;
      }
    }
  }
}

extern "C"
__global__
void viterbi_forward(
    const obs_t* __restrict__ obs, const int* __restrict__ obs_lens, int maxlen,
    prob_t* __restrict__ chi1, prob_t* __restrict__ chi2,
    state_t* __restrict__ zeta, int t, int k, HMM_DECL_PARAMS) {
  state_t state = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int instance = blockIdx.y;
  if (state < NUM_STATES) {
    prob_t *chi_prev, *chi_curr;
    if (k % 2 == 0) {
      chi_prev = chi2;
      chi_curr = chi1;
    } else {
      chi_prev = chi1;
      chi_curr = chi2;
    }
    size_t idx = instance * NUM_STATES + state;
    size_t zeta_idx = instance * BATCH_SIZE * NUM_STATES + (k-1) * NUM_STATES + state;
    if (t+k < obs_lens[instance]) {
      obs_t x = obs[instance * maxlen + t + k];
      state_t maxs;
      prob_t maxp = -inf;
      viterbi_max_predecessor(chi_prev, instance, state, &maxs, &maxp, HMM_CALL_ARGS);
      chi_curr[idx] = maxp + output_prob(state, x, HMM_CALL_ARGS);
      zeta[zeta_idx] = maxs;
    } else if (t+k == obs_lens[instance]) {
      // We only need to copy over chi data once - past this point, we know
      // both chi vectors will contain identical information. We continue
      // setting the zeta matrix as below to ensure we backtrack through it
      // correctly.
      chi_curr[idx] = chi_prev[idx];
      zeta[zeta_idx] = state;
    } else {
      zeta[zeta_idx] = state;
    }
  }
}

__device__
void viterbi_backward_warp_reduce(volatile prob_t *maxp, volatile state_t *maxs, unsigned int tid) {
  if (maxp[tid + 32] > maxp[tid]) {
    maxp[tid] = maxp[tid + 32];
    maxs[tid] = maxs[tid + 32];
  }
  if (maxp[tid + 16] > maxp[tid]) {
    maxp[tid] = maxp[tid + 16];
    maxs[tid] = maxs[tid + 16];
  }
  if (maxp[tid + 8] > maxp[tid]) {
    maxp[tid] = maxp[tid + 8];
    maxs[tid] = maxs[tid + 8];
  }
  if (maxp[tid + 4] > maxp[tid]) {
    maxp[tid] = maxp[tid + 4];
    maxs[tid] = maxs[tid + 4];
  }
  if (maxp[tid + 2] > maxp[tid]) {
    maxp[tid] = maxp[tid + 2];
    maxs[tid] = maxs[tid + 2];
  }
  if (maxp[tid + 1] > maxp[tid]) {
    maxp[tid] = maxp[tid + 1];
    maxs[tid] = maxs[tid + 1];
  }
}

extern "C"
__global__
void viterbi_backward(
    const prob_t* __restrict__ chi, const state_t* __restrict__ zeta,
    state_t* __restrict__ out, int maxlen, int T) {
  size_t idx = threadIdx.x;
  size_t instance = blockIdx.x;
  size_t lo = instance * NUM_STATES;

  __shared__ state_t maxs[512];
  __shared__ prob_t maxp[512];
  maxs[idx] = idx;
  if (idx < NUM_STATES) {
    maxp[idx] = chi[lo + idx];
  } else {
    maxp[idx] = -inf;
  }
  for (int i = lo + idx + 512; i < lo + NUM_STATES; i += 512) {
    if (chi[i] > maxp[idx]) {
      maxp[idx] = chi[i];
      maxs[idx] = i - lo;
    }
  }
  __syncthreads();

  if (idx < 256) {
    if (maxp[idx + 256] > maxp[idx]) {
      maxp[idx] = maxp[idx + 256];
      maxs[idx] = maxs[idx + 256];
    }
  }
  __syncthreads();
  if (idx < 128) {
    if (maxp[idx + 128] > maxp[idx]) {
      maxp[idx] = maxp[idx + 128];
      maxs[idx] = maxs[idx + 128];
    }
  }
  __syncthreads();
  if (idx < 64) {
    if (maxp[idx + 64] > maxp[idx]) {
      maxp[idx] = maxp[idx + 64];
      maxs[idx] = maxs[idx + 64];
    }
  }
  __syncthreads();
  if (idx < 32) viterbi_backward_warp_reduce(maxp, maxs, idx);

  // Run the backtracking sequentially from the maximum state using the first
  // thread for each instance.
  if (idx == 0) {
    state_t max_state = maxs[0];
    state_t *outptr = out + instance * maxlen + T;
    const state_t *zetaptr = zeta + instance * BATCH_SIZE * NUM_STATES;
    outptr[BATCH_SIZE-1] = max_state;
    for (int t = BATCH_SIZE-2; t >= 0; t--) {
      outptr[t] = zetaptr[t * NUM_STATES + outptr[t+1]];
    }
  }
}
